#include "hip/hip_runtime.h"
/*
Title: mandelbrot.cu
Author: Justin Staples
Date: May 12, 2018
Usage: 
	nvcc mandelbrot.cu -lGL -lGLU -lglut -o prog
	./prog
*/

#include <stdio.h>
#include <GL/gl.h> // OpenGL as well as utilities
#include <GL/glu.h>
#include <GL/glut.h>
#include <thrust/complex.h> // for complex numbers in CUDA

using namespace thrust;

typedef struct {double real; double imag;} point; // complex numbers

int MAX_ITER = 500; 
int W = 1024; // image dimensions
int H = 512;
int aspect_ratio = W / H;
int block_dim_x = 4; // 4 * 4 = 16 threads per block
int block_dim_y = 4;
dim3 block(block_dim_x, block_dim_y); // block and grid dimsions
dim3 grid(W / block_dim_x, H / block_dim_y);
double height = 1;
double width = height * aspect_ratio;
double scale = 1;
point center;
point bottom_left;

double orbitReal = 0.0;
double orbitImaginary = 0.0;
double * orbitRealRef = &orbitReal;
double * orbitImaginaryRef = &orbitImaginary;

int * d_out;
int * h_out;
const int ARRAY_SIZE = W * H;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

// takes an integer respresenting the number of iterations and uses it to assign RGB values. RGBA data can be exactly stored in a 4-byte int
__device__ int assignColour(int i) {
	int val = 0;
	// val += powf(i, 1); // blue
	val = val << 8; // each integer is 4 bytes but each colour component is only 1, so shift them to the correct positions (RGBA)
	val += powf(i, 2); // green
	val = val << 8;
	// val += powf(i, 1); // red

	return val;
}

// given a complex point C and an upper bound, determine the number of iterations until divergence
__device__ int numberOfIterations(complex<double> orbit, complex<double> C, int max) {
	complex<double> z0 = orbit; // orbit point, for mandelbrot z0 = 0 + 0i
	complex<double> zn; // populated during iteration

	int iter = 0;
	double mag = 0;	
	while (mag < 2 && iter < max) { // ensure the magnitude is bounded by 2
		mag = abs(z0);
		zn = z0 * z0 + C;
		z0 = zn;
		iter++;
	}
	return iter;
}

// process each pixel/thread individually
__global__ void process(int * d_out, point p, double w, double h, double s, int max, complex<double> orbit){
	// map block and grid coordinates to pixel coordinates
	int y_idx = 4 * blockIdx.y + threadIdx.y;
	int x_idx = 4 * blockIdx.x + threadIdx.x;
	int idx = 1024 * y_idx + x_idx;

	// map pixel coordinates to real and imaginary components
	double R = p.real + (x_idx / 1023.0) * w;
	double I = p.imag + (y_idx / 511.0) * h;

	// determine number of iterations for each complex point, bounded the maximum number of iterations
	complex<double> C(R, I); // each thread is responsible for 1 complex point P
	int iter = numberOfIterations(orbit, C, max);
	
	// use number of iterations to assign colours to each point
	int val = assignColour(iter);

	// write the value to the device output
	d_out[idx] = val;
}

// callback function for drawing the scene
void display() {
    glClearColor(0, 0, 0, 1);
    glClear(GL_COLOR_BUFFER_BIT);

	complex<double> orbit(*orbitRealRef, *orbitImaginaryRef);
	process<<<grid, block>>>(d_out, bottom_left, width, height, scale, MAX_ITER, orbit);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    glDrawPixels(W, H, GL_RGBA, GL_UNSIGNED_BYTE, h_out);

    glutSwapBuffers();
    glutPostRedisplay();
}

// callback function for keyboard keys
void keyboard(unsigned char key, int x, int y) {
	switch (key) {
		case 'w':
			scale *= 1.07;
			height /= 1.07;
			width /= 1.07;
			bottom_left.real = center.real - width / 2.0;
			bottom_left.imag = center.imag - height / 2.0;
			break;
		case 's':
			scale /= 1.07;
			height *= 1.07;
			width *= 1.07;
			bottom_left.real = center.real - width / 2.0;
			bottom_left.imag = center.imag - height / 2.0;
			break;
		case 'i':
			orbitImaginary += 0.01;
			break;
		case 'k':
			orbitImaginary -= 0.01;
			break;
		case 'j':
			orbitReal -= 0.01;
			break;
		case 'l':
			orbitReal += 0.01;
			break;
	}
}

// callback function for special (arrow) keys
void special(int key, int x, int y) {
	switch (key) {
		case GLUT_KEY_RIGHT:
			center.real += width / 100;
			bottom_left.real += width / 100;
			break;
		case GLUT_KEY_LEFT:
			center.real -= width / 100;
			bottom_left.real -= width /100;
			break;
		case GLUT_KEY_UP:
			center.imag += width / 100;
			bottom_left.imag += width / 100;
			break;
		case GLUT_KEY_DOWN:
			center.imag -= width / 100;
			bottom_left.imag -= width / 100;
			break;
	}
	glutSwapBuffers();
}

int main(int argc, char **argv)	{
	// initialize globals
	height = 2;
	width = 4;
	center.real = 0.0;
	center.imag = 0.0;
	bottom_left.real = center.real - width / 2.0;
	bottom_left.imag = center.imag - height / 2.0;

	// allocate memory on the CPU for host data and on the GPU for device data
	h_out = (int *) malloc(ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	// initialize GLUT and start the main program
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(W, H);
    glutCreateWindow("Mandelbrot");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutSpecialFunc(special);
    glutMainLoop();

    return 0;
}