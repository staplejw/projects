#include "hip/hip_runtime.h"
/*
Title: mandelbrot.cu
Author: Justin Staples
Date: May 12, 2018
Usage: 
	nvcc mandelbrot.cu -lGL -lGLU -lglut -o prog
	./prog
*/

#include <stdio.h>
#include <GL/gl.h> // OpenGL as well as utilities
#include <GL/glu.h>
#include <GL/glut.h>
#include <thrust/complex.h> // for complex numbers in CUDA

using namespace thrust;

typedef struct {float real; float imag;} point; // complex numbers

int MAX_ITER = 255; 
int W = 1024; // image dimensions
int H = 512;
int aspect_ratio = W / H;
int block_dim_x = 4; // 4 * 4 = 16 threads per block
int block_dim_y = 4;
dim3 block(block_dim_x, block_dim_y); // block and grid dimsions
dim3 grid(W / block_dim_x, H / block_dim_y);
float height = 1;
float width = height * aspect_ratio;
float scale = 1;
point center;
point bottom_left;

int * d_out;
int * h_out;
const int ARRAY_SIZE = W * H;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

__device__ int assignColour(int i) {
	int val = 0;
	val += powf(i, 2); // blue
	val = val << 8; // each integer is 4 bytes but each colour component is only 1, so shift them to the correct positions (RGBA)
	val += powf(i, 2); // green
	val = val << 8;
	val += powf(i, 2); // red

	return val;
}

__device__ int iterations(complex<float> C, int max) {
	complex<float> z0(0, 0); //  mandelbrot set defined for z0 = 0 + 0i
	complex<float> zn; // populated during iteration

	int iter = 0;
	float mag = 0;	
	while (mag < 2 && iter < max) { // ensure the magnitude is bounded by 2
		mag = abs(z0);
		zn = z0 * z0 + C;
		z0 = zn;
		iter++;
	}
	return iter;
}

__global__ void process(int * d_out, point p, float w, float h, float s, int max){
	// map block and grid coordinates to pixel coordinates
	int y_idx = 4 * blockIdx.y + threadIdx.y;
	int x_idx = 4 * blockIdx.x + threadIdx.x;
	int idx = 1024 * y_idx + x_idx;

	// map pixel coordinates to real and imaginary components
	float R = p.real + (x_idx / 1023.0) * w;
	float I = p.imag + (y_idx / 511.0) * h;

	// determine number of iterations for each complex point, bounded the maximum number of iterations
	complex<float> C(R, I); // each thread is responsible for 1 complex point P
	int iter = iterations(C, max);
	
	// use number of iterations to assign colours to each point
	int val = assignColour(iter);

	// write the value to the device output
	d_out[idx] = val;
}

void display() {
    glClearColor(0, 0, 0, 1);
    glClear(GL_COLOR_BUFFER_BIT);

	process<<<grid, block>>>(d_out, bottom_left, width, height, scale, MAX_ITER);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    glDrawPixels(W, H, GL_RGBA, GL_UNSIGNED_BYTE, h_out);

    glutSwapBuffers();
    glutPostRedisplay();
}

void keyboard(unsigned char key, int x, int y) {
	switch (key) {
		case 'w':
			scale *= 1.07;
			height /= 1.07;
			width /= 1.07;
			bottom_left.real = center.real - width / 2.0;
			bottom_left.imag = center.imag - height / 2.0;
			break;
		case 's':
			scale /= 1.07;
			height *= 1.07;
			width *= 1.07;
			bottom_left.real = center.real - width / 2.0;
			bottom_left.imag = center.imag - height / 2.0;
			break;
	}
}

void special(int key, int x, int y) {
	switch (key) {
		case GLUT_KEY_RIGHT:
			center.real += width / 100;
			bottom_left.real += width / 100;
			break;
		case GLUT_KEY_LEFT:
			center.real -= width / 100;
			bottom_left.real -= width /100;
			break;
		case GLUT_KEY_UP:
			center.imag += width / 100;
			bottom_left.imag += width / 100;
			break;
		case GLUT_KEY_DOWN:
			center.imag -= width / 100;
			bottom_left.imag -= width / 100;
			break;
	}
	glutSwapBuffers();
}

int main(int argc, char **argv)	{
	// initialize globals
	height = 2;
	width = 4;
	center.real = 0.0;
	center.imag = 0.0;
	bottom_left.real = center.real - width / 2.0;
	bottom_left.imag = center.imag - height / 2.0;

	// allocate memory on the CPU for host data and on the GPU for device data
	h_out = (int *) malloc(ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	// initialize GLUT and start the main program
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(W, H);
    glutCreateWindow("Mandelbrot");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutSpecialFunc(special);
    glutMainLoop();

    return 0;
}